#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"  
#include "hipblas.h"  
#include <time.h>
#include <windows.h>
#include <iostream>  

using namespace std;

int gettimeofday(struct timeval *tp, void *tzp)
{
	time_t clock;
	struct tm tm;
	SYSTEMTIME wtm;
	GetLocalTime(&wtm);
	tm.tm_year = wtm.wYear - 1900;
	tm.tm_mon = wtm.wMonth - 1;
	tm.tm_mday = wtm.wDay;
	tm.tm_hour = wtm.wHour;
	tm.tm_min = wtm.wMinute;
	tm.tm_sec = wtm.wSecond;
	tm.tm_isdst = -1;
	clock = mktime(&tm);
	tp->tv_sec = clock;
	tp->tv_usec = wtm.wMilliseconds * 1000;
	return (0);
}

// ������Ծ����ά��  
int const M = 8192;
int const N = M;

extern void printMatrix(float* m_Matrix, int W, int H);

int main()
{
	// ����״̬����  
	hipblasStatus_t status;
	
	// �� �ڴ� ��Ϊ��Ҫ����ľ��󿪱ٿռ�  
	float *h_A = (float*)malloc(N*M * sizeof(float));
	float *h_B = (float*)malloc(N*M * sizeof(float));

	// �� �ڴ� ��Ϊ��Ҫ����������ľ��󿪱ٿռ�  
	float *h_C = (float*)malloc(M*M * sizeof(float));



	// Ϊ����������Ԫ�ظ��� 0-10 ��Χ�ڵ������  
	for (int i = 0; i < N*M; i++) {
		h_A[i] = i%3;
		h_B[i] = i%2;

	}

	printMatrix(h_A,2,2);
	cout << endl;
	printMatrix(h_B,2,2);
	hipSetDevice(0);
	/*
	** GPU ����������
	*/

	// ��������ʼ�� CUBLAS �����  
	hipblasHandle_t handle;
	status = hipblasCreate(&handle);

	if (status != HIPBLAS_STATUS_SUCCESS)
	{
		if (status == HIPBLAS_STATUS_NOT_INITIALIZED) {
			cout << "CUBLAS ����ʵ��������" << endl;
		}
		getchar();
		return EXIT_FAILURE;
	}

    struct timeval t1, t2;
    gettimeofday(&t1, NULL);
    double timeuse;

	time_t z1 = time(NULL);
	printf("t1 %d\n", z1);

	hipEvent_t start_device, stop_device;
	float time_device;
	hipEventCreate(&start_device);
	hipEventCreate(&stop_device);
	hipEventRecord(start_device, 0);
	clock_t start_devence = clock();

	float *d_A, *d_B, *d_C, *d_D;
	hipMalloc((void**)&d_A, N*M * sizeof(float));
	hipMalloc((void**)&d_B, N*M * sizeof(float));
	hipMalloc((void**)&d_C, N*M * sizeof(float));
	hipMalloc((void**)&d_D, N*M * sizeof(float));

	hipblasSetVector(N*M,sizeof(float),h_A,1,d_A,1);
	hipblasSetVector(N*M,sizeof(float),h_B,1,d_B,1);

	// ͬ������  
	hipDeviceSynchronize();

	// ���ݽ�������˺����еĲ��������庬����ο������ֲᡣ  
	float a = 1; float b = 0;
	// ������ˡ��ú�����Ȼ���������������������  
	hipblasSgemm(
		handle,    // blas �����   
		HIPBLAS_OP_T,    // ���� A ���Բ���  
		HIPBLAS_OP_T,    // ���� B ���Բ���  
		M,    // A, C ������   
		M,    // B, C ������  
		N,    // A �������� B ������  
		&a,    // ����ʽ�� �� ֵ  
		d_A,    // A ���Դ��еĵ�ַ  
		N,    // lda  
		d_B,    // B ���Դ��еĵ�ַ  
		M,    // ldb  
		&b,    // ����ʽ�� �� ֵ  
		d_C,    // C ���Դ��еĵ�ַ(�������)  
		M    // ldc  
	);

	// ͬ������  
	hipDeviceSynchronize();
	// �� �Դ� ��ȡ���������� �ڴ���ȥ  
	hipblasGetVector(
		M*M,    //  Ҫȡ��Ԫ�صĸ���  
		sizeof(float),    // ÿ��Ԫ�ش�С  
		d_C,    // GPU ����ʼ��ַ
		1,    // ����Ԫ��֮��Ĵ洢���  
		h_C,    // ��������ʼ��ַ  
		1    // ����Ԫ��֮��Ĵ洢���  
	);

	for (int i = 0; i < 4; i++) {
		h_C[i] = h_C[i*M];
	}
	printMatrix(h_C,2,2);


	hipEventRecord(stop_device, 0);
	hipEventSynchronize(stop_device);
	hipEventElapsedTime(&time_device, start_device, stop_device);
	hipEventDestroy(start_device);
	hipEventDestroy(stop_device);
	cout << "gputime=" << time_device << "ms" << endl;
	cout << "devence=" << (double)(clock() - start_devence) / 1 << "ms" << endl;

    gettimeofday(&t2, NULL);
    timeuse = t2.tv_sec - t1.tv_sec + (t2.tv_usec - t1.tv_usec) / 1000000.0;
    printf("Use Time:%f\n", timeuse);

	// �����ʹ�ù����ڴ�  
	free(h_A);
	free(h_B);
	free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// �ͷ� CUBLAS �����  
	hipblasDestroy(handle);
	time_t z2 = time(NULL);
	printf("t2 %d\n", z2);
	printf("time %d\n", z2-z1);
	getchar();

	return 0;
}
